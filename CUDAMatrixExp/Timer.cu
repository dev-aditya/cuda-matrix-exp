//
// Cardiff University | Computer Science
// Module:     CM3203 One Semester Project (40 Credits)
// Title:      Parallelisation of Matrix Exponentials in C++/CUDA for Quantum Control
// Date:       2016
//
// Author:     Peter Davison
// Supervisor: Dr. Frank C Langbein
// Moderator:  Dr. Irena Spasic
//

// Include header file
#include "Timer.cuh"

Timer::Timer() {
	hipEventCreate(start);
	hipEventCreate(stop);
}
Timer::~Timer() {
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
void Timer::start() {
	hipEventRecord(start, 0);
}
void Timer::stop() {
	hipEventRecord(stop, 0);
}
void Timer::print() {
	hipEventElapsedTime(&time, start, stop);
	std::cout << time;
}